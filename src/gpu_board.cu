#include "hip/hip_runtime.h"
#include <stdexcept>
#include <random>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "board.h"

__device__ int get_remainder_gpu(int dividend, int divisor) {
    return (dividend % divisor + divisor) % divisor;
}

std::vector<std::vector<int>> GPUBoard::getBoard() {
    std::vector<std::vector<int>> standardised_board;
    thrust::host_vector<int> tmp_host_board = board;
    for (int i = 0; i < size; i++) {
        std::vector<int> row;
        for (int j = 0; j < size; j++) {
            row.push_back(tmp_host_board[(i*size)+j]);
        }
        standardised_board.push_back(row);
    }
    return standardised_board;
}

void GPUBoard::setManualSeed(const std::vector<std::vector<int>>& seed_board) {
    for (auto row : seed_board) {
        for (auto cell : row) {
            board.push_back(cell);
        }
    }
}

void GPUBoard::setRandomSeed() {
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(0,1);
    thrust::host_vector<int> tmp_host_vector;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            tmp_host_vector.push_back(distribution(generator));
        }
    }
    board = tmp_host_vector;
}

__device__ int compute_alive_neighbour_cnt(const int x, const int y, const int size, const int* old_board) {
    int cnt_alive_neighbours = 0;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            int idx = (get_remainder_gpu(x - 1 + i, size)*size) + get_remainder_gpu(y - 1 + j, size);
            if ((old_board[idx] == 1) && (!(i == 1 && j == 1))) {
                cnt_alive_neighbours++;
            }
        }
    }
    return cnt_alive_neighbours;
}

__global__ void evolve_board(int size, int* board, int* old_board) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < size*size; i+=stride) {
        int row = i / size;
        int col = i % size;
        int alive_neighbour_cnt = compute_alive_neighbour_cnt(row, col, size, old_board);
        if (old_board[i] == 1 && alive_neighbour_cnt < 2) { // dies from underpopulation
            board[i] = 0;
        } else if (old_board[i] == 1 && alive_neighbour_cnt > 3) {// dies from overcrowding
            board[i] = 0;
        } else if (old_board[i] == 0 && alive_neighbour_cnt == 3) { // birth
            board[i] = 1;
        }
    }   
}

void GPUBoard::evolve() {
    generation++;
    int block_size= 256;
    int num_blocks= ((size*size) + block_size - 1) / block_size;
    //thrust::copy(board.begin(), board.end(), old_board.begin());
    old_board = board;
    evolve_board<<<num_blocks, block_size>>>(
        size, thrust::raw_pointer_cast(board.data()), thrust::raw_pointer_cast(old_board.data())
    );    
}